
#include <hip/hip_runtime.h>


#define BLK_SIZE    1024


__global__ void saxpy_cuda(int n, float a, float *x, float *y){
     int t_id = threadIdx.x + blockDim.x * blockIdx.x;
     int stride = blockDim.x * gridDim.x;
     for (int i = t_id; i < n; i += stride)
     {
          y[i] = a * x[i] + y[i];
     }
}

int main()
{
     long n = 1UL << 30;
     float *x, *y, alpha=2.0;
     float *dx, *dy;

     x = new float[n];
     y = new float[n];

     hipMalloc(&dx, n * sizeof(float));
     hipMalloc(&dy, n * sizeof(float));

     for(long i=0;i<n;i++){
          x[i] = 1.f;
          y[i] = 1.f;
     }

     hipMemcpy(dx, x, n*sizeof(float), hipMemcpyHostToDevice);
     hipMemcpy(dy, y, n*sizeof(float), hipMemcpyHostToDevice);

     // 2^(10*5)
     saxpy_cuda<<<(n+BLK_SIZE-1)/BLK_SIZE, BLK_SIZE>>>(n, alpha, dx, dy);
     hipDeviceSynchronize();

     hipMemcpy(y, dy, n*sizeof(float), hipMemcpyDeviceToHost);

     hipFree(dx);
     hipFree(dy);


     delete[] x;
     delete[] y;

     return 0;
}